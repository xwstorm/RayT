#include "hip/hip_runtime.h"

#include "scene.cuh"
#include "object.cuh"
#include "hip/hip_vector_types.h"
#include <iostream>

int initDeviceScene(int width, int height, CUSphere*& spheres, gvec3*& hostMap, gvec3*& colorMap) {
    const int sphereSize = 8;

    CUSphere* hostSphere = new CUSphere[8]{

        CUSphere(1e5, gvec3(1e5 + 1,40.8,81.6), gvec3(),           gvec3(.75,.25,.25),   REF_DIFF),
        CUSphere(1e5, gvec3(-1e5 + 99,40.8,81.6),gvec3(),           gvec3(.25,.25,.75),   REF_DIFF),
        CUSphere(1e5, gvec3(50,40.8, 1e5),     gvec3(),           gvec3(.75,.75,.75),   REF_DIFF),
        //CUObject(1e5, vec3d(50,40.8,-1e5+170), vec3d(),           vec3d(),              REF_DIFF),
        CUSphere(1e5, gvec3(50, 1e5, 81.6),    gvec3(),           gvec3(.75,.75,.75),   REF_DIFF),
        CUSphere(1e5, gvec3(50,-1e5 + 81.6,81.6),gvec3(),           gvec3(.75,.75,.75),   REF_DIFF),
        CUSphere(16.5,gvec3(27,16.5,47),       gvec3(),           gvec3(1,1,1)*.999,    REF_SPEC),
        CUSphere(16.5,gvec3(73,16.5,78),       gvec3(),           gvec3(1,1,1)*.999,    REF_REFR),
        CUSphere(600, gvec3(50,681.6 - .27,81.6),gvec3(12,12,12),   gvec3(),              REF_DIFF)
    };

    CUSphere* deviceSphere;
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&deviceSphere, sphereSize * sizeof(CUSphere));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(deviceSphere, hostSphere, sphereSize * sizeof(CUSphere), hipMemcpyHostToDevice);

    cudaStatus = hipMalloc((void**)&colorMap, width * height * sizeof(gvec3));
    cudaStatus = hipMemcpy(colorMap, hostMap, width * height * sizeof(gvec3), hipMemcpyHostToDevice);
    spheres = deviceSphere;
    return sphereSize;
Error:
    //hipFree(deviceSphere);
    return 0;
}



__global__ void sphereKernel(
    CUSphere* sphereArr,
    int sphereSize,
    CURay cam,
    int depth,
    int sample,
    int width,
    int height,
    gvec3 cx,
    gvec3 cy,
    hiprandState *const rngStates,
    gvec3* colorMap)
{
    printf("[%d %d %d]\n", blockIdx.x, threadIdx.x, blockDim.x);
    //return;
    int x = threadIdx.x;
    int y = blockIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    gvec3 red(255.0, 0.0, 0.0);
    //red = red * 0.9;
    colorMap[tid] = red * 0.7;
    //return;
    // ÿ������2x2�Ĵ�С�ϼ���
    gvec3 color;
    int i, sy;
    hiprandState localState = rngStates[threadIdx.x];
    for (int m = 0; m < GLOBAL_RATE; ++m)
    {
        int y = blockIdx.x * GLOBAL_RATE + m;
        if (y >= height)
        {
            continue;
        }
        for (int n = 0; n < GLOBAL_RATE; ++n)
        {
            int x = threadIdx.x * GLOBAL_RATE + n;
            if (x >= width)
            {
                continue;
            }
            for (sy = 0; sy < 2; sy++)     // 2x2 subpixel rows
            {
                for (int sx = 0; sx < 2; sx++) {        // 2x2 subpixel cols
                    gvec3 tmpRes;
                    gvec3 tmpvec;
                    for (int s = 0; s < sample; ++s)
                    {
                        tmpvec = red * 5.0;
                        //gvec3 tmpvec(5.0, 5.0, 5.0);
                        double r1 = 2 * hiprand_uniform_double(&localState);
                        double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
                        //double r2 = 2 * hiprand_uniform_double(&localState);
                        double r2 = r1;
                        double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);
                        //continue;
                        // �����Ǽ�����ߵķ���
                        // ���ü����x����-0.5��0.5��
                        // ���ü��浽����ľ�����1
                        //tmpRes.x = 1.0;
                        double value = (((sx + .5 + dx) / 2 + x) / width - 0.5);
                        double cxxx = cx.x;
                        double tx = cx.x * value;
                        gvec3 tmpDir;
                        tmpDir.x = tx;
                        continue;
                        //gvec3 tmpDir = red*value;
                        //gvec3 tmpDir(cx.x*value, cx.y*value, cx.z*value);
                        gvec3 dir = cx*(((sx + 0.5 + dx) / 2.0 + x) / width - 0.5) + cy*(((sy + .5 + dy) / 2.0 + y) / height - .5) + cam.dir;
                        dir = glm::normalize(dir);

                        CURay cuRay(cam.ori, dir);
                        gvec3 ret = radiance_device(sphereArr, sphereSize, cuRay, 0, &localState) * (1.0 / sample);
                        tmpRes += ret;
                    }
                    gvec3 pointValue = gvec3(clamp(tmpRes.x, 0.0f, 1.0f), clamp(tmpRes.y, 0.0f, 1.0f), clamp(tmpRes.z, 0.0f, 1.0f))*.25;//�������4
                    color += pointValue;
                }
            }
            int index = y * width + x;
            colorMap[index] = red;
        }
    }

}



__global__ void sphereKernelTest(
    CUSphere* sphereArr,
    int sphereSize,
    CURay cam,
    int depth,
    int sample,
    int width,
    int height,
    gvec3 cx,
    gvec3 cy,
    hiprandState *const rngStates,
    gvec3* colorMap)
{
    printf("[%d %d %d]\n", blockIdx.x, threadIdx.x, blockDim.x);
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    gvec3 red(255.0, 0.0, 0.0);
    // ÿ������2x2�Ĵ�С�ϼ���
    hiprandState localState = rngStates[tid];
    int index = blockIdx.x * THREAD_DIM + threadIdx.x;
    int yy = (index / OUT_WIDTH) * HEIGHT_STEP;
    int x = index % OUT_WIDTH;

    for (int i = 0; i < HEIGHT_STEP; ++i)
    {
        int y = yy + i;
        gvec3 color;
        for (int sy = 0; sy < 2; sy++)     // 2x2 subpixel rows
        {
            for (int sx = 0; sx < 2; sx++) // 2x2 subpixel cols
            {        
                gvec3 tmpRes;
                for (int s = 0; s < sample; ++s)
                {
                    double r1 = 2 * hiprand_uniform_double(&localState);
                    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
                    double r2 = 2 * hiprand_uniform_double(&localState);
                    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);
                    // �����Ǽ�����ߵķ���
                    // ���ü����x�Ǵ�-0.5��0.5��
                    // ���ü��浽����ľ�����1
                    gvec3 dir = cx*(((sx + 0.5 + dx) / 2.0 + x) / width - 0.5) + cy*(((sy + 0.5 + dy) / 2.0 + y) / height - 0.5) + cam.dir;
                    dir = glm::normalize(dir);
                    CURay cuRay(cam.ori, dir);
                    gvec3 ret = radiance_device(sphereArr, sphereSize, cuRay, 0, &localState) * (1.0 / sample);
                    //gvec3 ret;
                    tmpRes += ret;
                    //tmpRes.x = r1;
                    //tmpRes.y = r2;
                }
                gvec3 pointValue = gvec3(clamp(tmpRes.x, 0.0f, 1.0f), clamp(tmpRes.y, 0.0f, 1.0f), clamp(tmpRes.z, 0.0f, 1.0f))*0.25;//�������4
                color += pointValue;
            }
        }
        int mapIndex = y * OUT_WIDTH + x;
        colorMap[mapIndex] = color;
    }
}



static __global__ void rngSetupStates(
    hiprandState *rngState,
    int device_id)
{
    // determine global thread id
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0, &rngState[tid]);
    hiprand_init(blockIdx.x, threadIdx.x, 0, &rngState[tid]);
}

__global__ void testWhy() {
    printf("testWhy [%d %d]\n", blockIdx.x, threadIdx.x);
}

void testKernel() {
    testWhy << <2, 2 >> > ();
    hipDeviceSynchronize();
}

void startKernel(
    CUSphere* sphereArr,
    int sphereSize,
    CURay& cam,
    int depth,
    int sample,
    int width,
    int height,
    gvec3& cx,
    gvec3& cy,
    hiprandState *const rngStates,
    int sample_count,
    gvec3* hostMap,
    gvec3* colorMap) {
    //testWhy << <2, 2 >> > ();
    //return;
    hipError_t cudaStatus = hipMalloc((void **)&rngStates, height * width * sizeof(hiprandState));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "error" << std::endl;
        return;
    }
    int blockDim = (height/ GLOBAL_RATE) + 1;
    int threadDim = (width/ GLOBAL_RATE) + 1;
    printf("begin set up rand state\n");
    rngSetupStates << <BLOCK_DIM, THREAD_DIM >> > (rngStates, 0);
    //rngSetupStates << <height, width >> > (rngStates, 0);
    checkCudaStatus();
    testWhy << <2, 2 >> > ();
    checkCudaStatus();
    hipDeviceSynchronize();
    checkCudaStatus();
    cudaThreadSetLimit(hipLimitStackSize, 1024 * 48);

    printf("begin kernel test\n");
    sphereKernelTest << <BLOCK_DIM, THREAD_DIM >> > (sphereArr, sphereSize, cam, 0, sample_count, width, height, cx, cy, rngStates, colorMap);
    checkCudaStatus();
    printf("begin synchronize\n");
    hipDeviceSynchronize();
    checkCudaStatus();
    hipError_t copyStatus = hipMemcpy(hostMap, colorMap, width * height * sizeof(gvec3), hipMemcpyDeviceToHost);
    if (copyStatus != hipSuccess)
    {
        printf("cuda error");
    }
}